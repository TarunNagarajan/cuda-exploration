#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *A, int *B, int *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(int);

    int *h_A = new int[N];
    int *h_B = new int[N];
    int *h_C = new int[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size); 
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize;

    vectorAdd<<<gridsize, blocksize>>>(d_A, d_B, d_C, N); 
    hipMemcpy(h_C, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        std::cout << "C[" << i << "] = " << h_C << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
